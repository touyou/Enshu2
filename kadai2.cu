#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#define TIMEMAX 100
#define XSIZE 50
#define YSIZE 50

/*__global__ void add(float *x, float *y, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    while (idx < n) {
        y[idx] = y[idx] + x[idx];
        idx = blockDim.x * gridDim.x + idx;
    }
}*/

// blockIdx, blockDim, threadIdx, gridDim
__global__ void simmGpu(float ***u, float r) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx == 0 || idy == 0 || idx == XSIZE-1 || idy == YSIZE-1) return;

    for (t=0; t<TIMEMAX; t++) {
        float u1 = u[t%2][idx][idy];
        float u2 = u[t%2][idx+1][idy];
        float u3 = u[t%2][idx-1][idy];
        float u4 = u[t%2][idx][idy+1];
        float u5 = u[t%2][idx][idy-1];

        __syncthreads();
        u[(t+1)%2][idx][idy] = (1.0 - 4.0*r) * u1 + r * (u2 + u3 + u4 + u5);
        __syncthreads();
    }
}

void simmCpu(float u[2][XSIZE][YSIZE], float r) {
    int t, i, j;
    omp_set_num_threads(8);

    for (t=0; t<TIMEMAX; t++) {
        #pragma omp parallel for
        for (i=1; i<XSIZE-1; i++) {
            #pragma omp parallel for
            for (j=1; j<YSIZE-1; j++) {
                u[(t+1)%2][i][j] = (1.0 - 4.0*r) * u[t%2][i][j] + r * (u[t%2][i+1][j] + u[t%2][i-1][j] + u[t%2][i][j+1] + u[t%2][i][j-1]);
            }
        }
    }
}

int divRoundUp(int value, int radix) {
    return (value + radix - 1) / radix;
}

int main() {
    struct timeval t0, t1;
    float ***devA;
    float u[2][XSIZE][YSIZE];
    int nb = 2 * XSIZE * YSIZE * sizeof(float), i, j;
    memset(u, 0, nb);
    for (i=1; i<XSIZE-1; i++) {
        for (j=1; j<YSIZE-1; j++) {
            u[0][i][j] = 1;
        }
    }

    gettimeofday(&t0, NULL);
    // simmCpu(u, 0.12);
    hipMalloc((void****)&devA, nb);
    hipMemcpy(devA, u, nb, hipMemcpyHostToDevice);
    simmGpu<<<100, XSIZE*YSIZE>>>(devA, 0.12);
    hipMemcpy(u, devA, nb, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    gettimeofday(&t1, NULL);
    printf("Elapsed time = %lf\n", (double)(t1.tv_sec-t0.tv_sec)+(double)(t1.tv_usec-t0.tv_usec)*1.0e-6);

    for (i=0; i<XSIZE; i++) {
        for (j=0; j<YSIZE; j++) {
            if (u[0][i][j] > 0.34) printf("#");
            else if (u[0][i][j] > -0.34) printf("*");
            else printf(".");
        }
        puts("");
    }
}
